
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>

using namespace std;

#define N 4096
#define RADIUS 3
#define BLOCK_SIZE 16

__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
      temp[lindex - RADIUS] = in[gindex - RADIUS];
      temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
      result += temp[lindex + offset];

    // Store the result
    out[gindex] = result;
}

void fill_ints(int *x, int n) {
  fill_n(x, n, 1);
}

int main(void) {
  int *in, *out; // host copies of a, b, c
  int *d_in, *d_out; // device copies of a, b, c

  // Alloc space for host copies and setup values
  int size = (N + 2 * RADIUS) * sizeof(int);
  in = (int *)malloc(size); fill_ints(in, N + 2*RADIUS);
  out = (int *)malloc(size); fill_ints(out, N + 2*RADIUS);

  // Alloc space for device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

  // Launch stencil_1d() kernel on GPU
  // 如果你不加上 RADIUS，线程中内核函数可能会从 d_in[0] 开始访问， 此时位于RADIUS上，这会导致在计算过程中访问到无效的边界元素，从而引发错误或未定义行为。所以需要加上指针偏移，让他从后面开始索引。
  stencil_1d<<<N/BLOCK_SIZE,BLOCK_SIZE>>>(d_in + RADIUS, d_out+ RADIUS);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  // Error Checking
  for (int i = 0; i < N + 2*RADIUS; i++) {
    if (i<RADIUS || i>=N+RADIUS){
      if (out[i] != 1)
    	printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1);
    } else {
      if (out[i] != 1 + 2*RADIUS)
    	printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1 + 2*RADIUS);
    }
  }

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  printf("Success!\n");
  return 0;
}
